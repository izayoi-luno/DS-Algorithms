#include <iostream>
#include <hip/hip_runtime.h>

__global__ void conv2d_cuda(float* input, float* kernel, float* output, int input_height, int input_width, int kernel_height, int kernel_width, int output_height, int output_width) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if (tx < output_width && ty < output_height) {
        float sum = 0.0f;
        for (int ky = 0; ky < kernel_height; ky++) {
            for (int kx = 0; kx < kernel_width; kx++) {
                int ix = tx + kx;
                int iy = ty + ky;
                sum += input[iy * input_width + ix] * kernel[ky * kernel_width + kx];
            }
        }
        output[ty * output_width + tx] = sum;
    }
}

int main() {
    const int input_height = 5, input_width = 5;
    const int kernel_height = 3, kernel_width = 3;
    const int output_height = input_height - kernel_height + 1;
    const int output_width = input_width - kernel_width + 1;

    // Allocate memory for input, kernel, and output
    float input[input_height][input_width] = {
        {1, 2, 3, 4, 5},
        {6, 7, 8, 9, 10},
        {11, 12, 13, 14, 15},
        {16, 17, 18, 19, 20},
        {21, 22, 23, 24, 25}
    };
    float kernel[kernel_height][kernel_width] = {
        {1, 0, -1},
        {1, 0, -1},
        {1, 0, -1}
    };
    float output[output_height][output_width] = {0};

    float *d_input, *d_kernel, *d_output;

    // Allocate device memory
    hipMalloc(&d_input, input_height * input_width * sizeof(float));
    hipMalloc(&d_kernel, kernel_height * kernel_width * sizeof(float));
    hipMalloc(&d_output, output_height * output_width * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, input, input_height * input_width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_height * kernel_width * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block size
    dim3 blockDim(16, 16);
    dim3 gridDim((output_width + blockDim.x - 1) / blockDim.x, (output_height + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    conv2d_cuda<<<gridDim, blockDim>>>(d_input, d_kernel, d_output, input_height, input_width, kernel_height, kernel_width, output_height, output_width);

    // Copy result back to host
    hipMemcpy(output, d_output, output_height * output_width * sizeof(float), hipMemcpyDeviceToHost);

    // Print output
    for (int i = 0; i < output_height; i++) {
        for (int j = 0; j < output_width; j++) {
            std::cout << output[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Free memory
    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);

    return 0;
}